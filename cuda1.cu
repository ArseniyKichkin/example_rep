#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"


__global__ void matrixMultiplication(float *A, float *B, float *C, int N) {
    const int TILE_SIZE = 32;
    __shared__ float A_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float B_tile[TILE_SIZE][TILE_SIZE];
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0;

    for (int k = 0; k < N/TILE_SIZE; k++) {
        A_tile[threadIdx.y][threadIdx.x] = A[row*N + (k*TILE_SIZE + threadIdx.x)];
        B_tile[threadIdx.y][threadIdx.x] = B[(k*TILE_SIZE+threadIdx.y)*N + col];
        __syncthreads();

        for (int i = 0; i < TILE_SIZE; i++) {
            sum += A_tile[threadIdx.y][i] * B_tile[i][threadIdx.x];
        }
        __syncthreads();
    }
    C[row*N+col] = sum;
}

int main() {
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float *A = (float*) malloc(size);
    float *B = (float*) malloc(size);
    float *C = (float*) malloc(size);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    matrixMultiplication<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
