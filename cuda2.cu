#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define TILE_SIZE 2

__global__ void matrixMultiplication(float *A, float *B, float *C, int N) {
//    __shared__ float A_tile[TILE_SIZE][TILE_SIZE];
//    __shared__ float B_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float A_tile[TILE_SIZE * TILE_SIZE];
    __shared__ float B_tile[TILE_SIZE * TILE_SIZE];
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0;

    for (int k = 0; k < N/TILE_SIZE; k++) {
//        A_tile[threadIdx.y][threadIdx.x] = A[row*N + (k*TILE_SIZE + threadIdx.x)];
//        B_tile[threadIdx.y][threadIdx.x] = B[(k*TILE_SIZE+threadIdx.y)*N + col];
          A_tile[threadIdx.y * TILE_SIZE + threadIdx.x] = A[row*N + (k*TILE_SIZE + threadIdx.x)];
          B_tile[threadIdx.y * TILE_SIZE + threadIdx.x] = B[(k*TILE_SIZE+threadIdx.y)*N + col];

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; i++) {
//            sum += A_tile[threadIdx.y][i] * B_tile[i][threadIdx.x];
              sum += A_tile[threadIdx.y * TILE_SIZE  + i] * B_tile[i * N + threadIdx.y];
        }
        __syncthreads();
    }
    C[row*N+col] = sum;
}
clock_t start, end;
float time_used;

int main() {
    int N = 4;
    size_t size = N * N * sizeof(float);

    float *A = (float*) malloc(size);
    float *B = (float*) malloc(size);
    float *C = (float*) malloc(size);
    for (int row = 0; row < N; ++row){
        for (int col = 0; col < N; ++col){
            A[N * row + col] = col + N * row;
            B[N * row + col] = 2. * (col + N * row);
        }
    }
    for (int row = 0; row < N; ++row){
        printf("%f %f %f %f\n", A[row*N], A[row*N + 1], A[row*N + 2], A[row*N + 3]);
    }
    printf("\n");
    for (int row = 0; row < N; ++row){
        printf("%f %f %f %f\n", B[row*N], B[row*N + 1], B[row*N + 2], B[row*N + 3]);
    }
    printf("\n");



    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);



    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
    start = clock();
    matrixMultiplication<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    end = clock();
    time_used = ((float) (end - start)) / CLOCKS_PER_SEC;

    for (int row = 0; row < N; ++row){
        printf("%f %f %f %f\n", C[row*N], C[row*N + 1], C[row*N + 2], C[row*N + 3]);
    }
    printf("%f\n", time_used);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}


